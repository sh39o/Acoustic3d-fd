//a#########################################################
//a##         3D Acoustic Isotropic Medium Forward
//a##    
//a##  Ps :GPU(CUDA)  
//a##
//a##/*a***************************
//a##Function for Isotropic medium modeling,
//a##
//a## Ps:  the function of modeling following:
//a##      
//a##          du/dt=1/rho*dp/dx , 
//a##          dv/dt=1/rho*dp/dy , 
//a##          dw/dt=1/rho*dp/dz ,  
//a##          dp/dt=rho*vp^2*(du/dx+dv/dy+dw/dz)
//a##  
//a##*********a*******************/
//a##
//a##                                  code by Rong Tao 
//a##                            
//a#########################################################
#include<stdio.h>
#include<malloc.h>
#include<math.h>
#include<stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define pi 3.141592653

#define BlockSize1 16// tile size in 1st-axis
#define BlockSize2 16// tile size in 2nd-axis

#define mm 4

__device__ float d0;

__constant__ float c[mm] = {1.196289, -0.0797526, 0.009570313, -0.0006975447};

//a################################################################################
void check_gpu_error(const char *msg)
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        printf("Cuda error: %s: %s\n", msg, hipGetErrorString(err));
        exit(0);
    }
}

//a################################################################################
__global__ void
add_source(float pfac, int fsx, int fsy, int sz, int nx, int ny, int nz, int nnx, int nny, int nnz, float dt, float t,
           float favg, int wtype, int npml, int is, int dsx, int dsy, float *P, int nsx)
/*< generate ricker wavelet with time deley >*/
{
    int ixs, iys, izs;
    float x_, xx_, tdelay, ts, source = 0.0, sx, sy;

    tdelay = 1.0 / favg;
    ts = t - tdelay;

    sx = fsx + is % nsx * dsx;
    sy = fsy + is / nsx * dsy;

    if (wtype == 1)//ricker wavelet
    {
        x_ = favg * ts;
        xx_ = x_ * x_;
        source = (1 - 2 * pi * pi * (xx_)) * exp(-(pi * pi * xx_));
    } else if (wtype == 2) {//derivative of gaussian
        x_ = (-4) * favg * favg * pi * pi / log(0.1);
        source = (-2) * pi * pi * ts * exp(-x_ * ts * ts);
    } else if (wtype == 3) {//derivative of gaussian
        x_ = (-1) * favg * favg * pi * pi / log(0.1);
        source = exp(-x_ * ts * ts);
    } else if (wtype == 4){
        x_ = exp( - favg * favg * pi * pi * ts * ts ) * t;
        source = -x_;
        
    }
    

    if (t <= 2 * tdelay) {
        ixs = sx + npml - 1;
        iys = sy + npml - 1;
        izs = sz + npml - 1;
        P[izs + ixs * nnz + iys * nnz * nnx] += pfac * source;
    }
}

/*******************func*********************/
__global__ void
update_vel(int nx, int ny, int nz, int nnx, int nny, int nnz, int npml, float dt, float dx, float dy, float dz,
           float *u0, float *v0, float *w0, float *u1, float *v1, float *w1, float *P,
           float *coffx1, float *coffx2, float *coffy1, float *coffy2, float *coffz1, float *coffz2, float *rho) {
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

    int id, iy, im;
    float dtx, dty, dtz, xx, yy, zz;

    dtx = dt / dx;
    dty = dt / dy;
    dtz = dt / dz;

    for (iy = 0; iy < nny; iy++) {
        id = iz + ix * nnz + iy * nnz * nnx;
        if (id >= mm && id < nnx * nny * nnz - mm) {
            if (ix >= mm && ix < (nnx - mm) && iy >= mm && iy < (nny - mm) && iz >= mm && iz < (nnz - mm)) {
                xx = 0.0;
                yy = 0.0;
                zz = 0.0;
                for (im = 0; im < mm; im++) {
                    yy += c[im] * (P[id + (im + 1) * nnz * nnx] - P[id - im * nnz * nnx]);
                    xx += c[im] * (P[id + (im + 1) * nnz] - P[id - im * nnz]);
                    zz += c[im] * (P[id + im + 1] - P[id - im]);
                }
                xx /= rho[id];
                yy /= rho[id];
                zz /= rho[id];
                u1[id] = coffx2[ix] * u0[id] - coffx1[ix] * dtx * xx;
                v1[id] = coffy2[iy] * v0[id] - coffy1[iy] * dty * yy;
                w1[id] = coffz2[iz] * w0[id] - coffz1[iz] * dtz * zz;
            }
        }
    }


}

/*******************func***********************/
__global__ void update_stress(int nx, int ny, int nz, int nnx, int nny, int nnz, float dt, float dx, float dy, float dz,
                              float *u1, float *v1, float *w1, float *P, float *vp, float *rho, int npml,
                              float *px1, float *px0, float *py1, float *py0, float *pz1, float *pz0,
                              float *acoffx1, float *acoffx2, float *acoffy1, float *acoffy2, float *acoffz1,
                              float *acoffz2,
                              int fsx, int dsx, int fsy, int dsy, int zs, int is, int nsx) {
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

    int id, iy, im;
    float dtx, dty, dtz, xx, yy, zz;

    dtx = dt / dx;
    dty = dt / dy;
    dtz = dt / dz;

    for (iy = 0; iy < nny; iy++) {
        id = iz + ix * nnz + iy * nnz * nnx;
        if (id >= mm && id < nnx * nnz * nny - mm) {
/************************i****************************************/
/************************iso circle start*************************/

/************************ iso circle end *************************/
/************************i****************************************/
            if (ix >= mm && ix < (nnx - mm) && iy >= mm && iy < (nny - mm) && iz >= mm && iz < (nnz - mm)) {
                xx = 0.0;
                yy = 0.0;
                zz = 0.0;
                for (im = 0; im < mm; im++) {
                    yy += c[im] * (v1[id + im * nnz * nnx] - v1[id - (im + 1) * nnz * nnx]);
                    xx += c[im] * (u1[id + im * nnz] - u1[id - (im + 1) * nnz]);
                    zz += c[im] * (w1[id + im] - w1[id - im - 1]);
                }
                px1[id] = acoffx2[ix] * px0[id] - acoffx1[ix] * rho[id] * vp[id] * vp[id] * dtx * xx;
                py1[id] = acoffy2[iy] * py0[id] - acoffy1[iy] * rho[id] * vp[id] * vp[id] * dty * yy;
                pz1[id] = acoffz2[iz] * pz0[id] - acoffz1[iz] * rho[id] * vp[id] * vp[id] * dtz * zz;

                P[id] = px1[id] + py1[id] + pz1[id];
            }
        }
    }
}

/********************func**********************/
__global__ void get_d0(float dx, float dy, float dz, int nnx, int nny, int nnz, int npml, float *vp) {
    d0 = 10.0 * vp[nny * nnx * nnz / 2] * log(100000.0) / (2.0 * npml * ((dx + dy + dz) / 3.0));
}

/*************func*******************/
void pad_vv(int nx, int ny, int nz, int nnx, int nny, int nnz, int npml, float *ee) {
    int ix, iy, iz, id;

    for (iy = 0; iy < nny; iy++)
        for (ix = 0; ix < nnx; ix++) {
            for (iz = 0; iz < nnz; iz++) {
                id = iz + ix * nnz + iy * nnz * nnx;

                if (ix < npml) {
                    ee[id] = ee[iz + npml * nnz + iy * nnz * nnx];  //left
                } else if (ix >= nnx - npml) {
                    ee[id] = ee[iz + (nnx - npml - 1) * nnz + iy * nnz * nnx];//right
                }
            }
        }
    for (iy = 0; iy < nny; iy++)
        for (ix = 0; ix < nnx; ix++) {
            for (iz = 0; iz < nnz; iz++) {
                id = iz + ix * nnz + iy * nnz * nnx;

                if (iy < npml) {
                    ee[id] = ee[iz + ix * nnz + npml * nnz * nnx];  //front
                } else if (iy >= nny - npml) {
                    ee[id] = ee[iz + ix * nnz + (nny - npml - 1) * nnz * nnx];//back
                }
            }
        }
    for (iy = 0; iy < nny; iy++)
        for (ix = 0; ix < nnx; ix++) {
            for (iz = 0; iz < nnz; iz++) {
                id = iz + ix * nnz + iy * nnz * nnx;

                if (iz < npml) {
                    ee[id] = ee[npml + ix * nnz + iy * nnz * nnx];  //up
                } else if (iz >= nnz - npml) {
                    ee[id] = ee[nnz - npml - 1 + ix * nnz + iy * nnz * nnx];//down
                }
            }
        }

}

/*************func*******************/
void
read_file(char FN1[], char FN4[], int nx, int ny, int nz, int nnx, int nny, int nnz, float *vv, float *rho, int npml) {
    int ix, iy, iz, id;

    FILE *fp1, *fp4;
    if ((fp1 = fopen(FN1, "rb")) == NULL)printf("error open <%s>!\n", FN1);
    if ((fp4 = fopen(FN4, "rb")) == NULL)printf("error open <%s>!\n", FN4);

    for (iy = npml; iy < ny + npml; iy++) {
        for (ix = npml; ix < nx + npml; ix++) {
            for (iz = npml; iz < nz + npml; iz++) {
                id = iz + ix * nnz + iy * nnz * nnx;
                fread(&vv[id], 4L, 1, fp1);//vv[id]=3000.0;
                fread(&rho[id], 4L, 1, fp4);//rho[id]=1.5;
            }
        }
    }
    fclose(fp1);
    fclose(fp4);
}

/*************func*******************/
__global__ void initial_coffe(float dt, int nn, float *coff1, float *coff2, float *acoff1, float *acoff2, int npml) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if (id < nn + 2 * npml) {
        if (id < npml) {
            coff1[id] = 1.0 / (1.0 + (dt * d0 * pow((npml - 0.5 - id) / npml, 2.0)) / 2.0);
            coff2[id] = coff1[id] * (1.0 - (dt * d0 * pow((npml - 0.5 - id) / npml, 2.0)) / 2.0);

            acoff1[id] = 1.0 / (1.0 + (dt * d0 * pow(((npml - id) * 1.0) / npml, 2.0)) / 2.0);
            acoff2[id] = acoff1[id] * (1.0 - (dt * d0 * pow(((npml - id) * 1.0) / npml, 2.0)) / 2.0);

        } else if (id >= npml && id < npml + nn) {

            coff1[id] = 1.0;
            coff2[id] = 1.0;

            acoff1[id] = 1.0;
            acoff2[id] = 1.0;

        } else {

            coff1[id] = 1.0 / (1.0 + (dt * d0 * pow((0.5 + id - nn - npml) / npml, 2.0)) / 2.0);
            coff2[id] = coff1[id] * (1.0 - (dt * d0 * pow((0.5 + id - nn - npml) / npml, 2.0)) / 2.0);

            acoff1[id] = 1.0 / (1.0 + (dt * d0 * pow(((id - nn - npml) * 1.0) / npml, 2.0)) / 2.0);
            acoff2[id] = acoff1[id] * (1.0 - (dt * d0 * pow(((id - nn - npml) * 1.0) / npml, 2.0)) / 2.0);
        }
    }
}

/*************func*******************/
__global__ void
shot_record(int nnx, int nny, int nnz, int nx, int ny, int nz, int npml, int it, int nt, float *P, float *shot) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    int ix = id % nx;
    int iy = id / nx;

    if (id < nx * ny) {
        shot[it + nt * ix + nt * nx * iy] = P[npml + nnz * (ix + npml) + nnz * nnx * (iy + npml)];
    }
}

/*************func**************/
void window3d(float *a, float *b, int nz, int nx, int ny, int nnz, int nnx, int npml)
/*< window a 3d subvolume >*/
{
    int iz, ix, iy;

    for (iy = 0; iy < ny; iy++) {
        for (ix = 0; ix < nx; ix++) {
            for (iz = 0; iz < nz; iz++) {
                a[iz + nz * ix + nz * nx * iy] = b[(iz + npml) + nnz * (ix + npml) + nnz * nnx * (iy + npml)];
            }
        }
    }
}

/*************func**************/
__global__ void
mute_directwave(int nx, int ny, int nt, float dt, float favg, float dx, float dy, float dz, int fsx, int fsy, int dsx,
                int dsy,
                int zs, int is, float *vp, float *shot, int tt, int nsx) {

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int id, it;
    int mu_t, mu_nt;
    float mu_x, mu_y, mu_z, mu_t0;

    for (it = 0; it < nt; it++) {
        id = it + ix * nt + iy * nx * nt;
        if (ix < nx && iy < ny && it < nt) {
            mu_x = dx * abs(ix - fsx - (is % nsx) * dsx);
            mu_y = dy * abs(iy - fsy - (is / nsx) * dsy);
            mu_z = dz * zs;
            mu_t0 = sqrtf(pow(mu_x, 2) + pow(mu_y, 2) + pow(mu_z, 2)) / (vp[1]);
            mu_t = (int) (2.0 / (dt * favg));
            mu_nt = (int) (mu_t0 / dt) + mu_t + tt;

            if (it < mu_nt)
                shot[id] = 0.0;
        }
    }
/*    int id=threadIdx.x+blockDim.x*blockIdx.x;

    int mu_t,mu_nt;
    float mu_x,mu_y,mu_z,mu_t0;

    int ix=(id/nt)%nx;
    int iy=(id/nt)/nx;
    int it=id%nt;

   if(id<nx*ny*nt)
   {
        mu_x=dx*abs(ix-fsx-(is%nsx)*dsx);
        mu_y=dy*abs(iy-fsy-(is/nsx)*dsy);
        mu_z=dz*zs;
        mu_t0=sqrtf(pow(mu_x,2)+pow(mu_y,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilon[1]));
        mu_t=(int)(2.0/(dt*favg));
        mu_nt=(int)(mu_t0/dt)+mu_t+tt;

           if(it<mu_nt)
              shot[id]=0.0;
   }  */
}

//a########################################################################
extern "C" void cuda_3dfd_1order(char *FNvel, char *FNrho, char *FNsnap, char *FNshot, int is, int ns,
                     int nx, int ny, int nz, float dx, float dy, float dz,
                     int sxbeg, int sybeg, int szbeg, int jsx, int jsy, int jsz,
                     float dgx, float dgy, float dgt,
                     int nt, float dt, float fm, bool show_snapshot, bool cut_directwave,
                     int snap_interval, int cudaDevicei){
                     
    int it, nnx, nny, nnz, wtype, ix, iy;
    int nsx, dsx, fsx, dsy, fsy, zs, npml;
    float t, pfac, favg;

    float *v, *e, *rho;
    float *vp, *density;
    float *s_u0, *s_u1, *s_px0, *s_px1;
    float *s_v0, *s_v1, *s_py0, *s_py1;
    float *s_w0, *s_w1, *s_pz0, *s_pz1;
    float *s_P, *shot_Dev, *shot_Hos, *ptr;

    float *coffx1, *coffx2, *coffy1, *coffy2, *coffz1, *coffz2;
    float *acoffx1, *acoffx2, *acoffy1, *acoffy2, *acoffz1, *acoffz2;

    hipError_t error;
/*************wavelet\boundary**************/
    wtype = 4;
    npml = 20;
/********** dat document ***********/
    char snapname[300], snapid[300];

/********aaa************/
    FILE *fpsnap, *fpshot;
    fpshot = fopen(FNshot, "wb");


/********* parameters *************/

    favg = fm;
    pfac = 10.0;

    nsx = ns;
    fsx = sxbeg;
    dsx = jsx;
    fsy = sybeg;
    dsy = jsy;
    zs = szbeg;
/*************v***************/
    nnx = nx + 2 * npml;
    nny = ny + 2 * npml;
    nnz = nz + 2 * npml;
/************a*************/


    v = (float *) malloc(nnz * nnx * nny * sizeof(float));
    e = (float *) malloc(nnz * nnx * nny * sizeof(float));
    rho = (float *) malloc(nnz * nnx * nny * sizeof(float));
    shot_Hos = (float *) malloc(nt * nx * ny * sizeof(float));
    read_file(FNvel, FNrho, nx, ny, nz, nnx, nny, nnz, v, rho, npml);
/****************************/

    pad_vv(nx, ny, nz, nnx, nny, nnz, npml, v);
    pad_vv(nx, ny, nz, nnx, nny, nnz, npml, rho);
    
    if(hipSetDevice(cudaDevicei) != hipSuccess){// initialize device, default device=0;
        printf("error in setting device\n");
        //check_gpu_error("Failed to initialize device!");
    }
    error=hipGetLastError();
    if(error != hipSuccess){
        printf("%s\n",hipGetErrorString(error));
    }

    dim3 Xdimg, dimg, dimb;
    Xdimg.x = (nnx + BlockSize1 - 1) / BlockSize1;
    Xdimg.y = (nny + BlockSize2 - 1) / BlockSize2;
    dimg.x = (nnz + BlockSize1 - 1) / BlockSize1;
    dimg.y = (nnx + BlockSize2 - 1) / BlockSize2;
    dimb.x = BlockSize1;
    dimb.y = BlockSize2;
/****************************/
    hipMalloc(&vp, nnz * nnx * nny * sizeof(float));
    hipMalloc(&density, nnz * nnx * nny * sizeof(float));
    hipMemcpy(vp, v, nnz * nnx * nny * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(density, rho, nnz * nnx * nny * sizeof(float), hipMemcpyHostToDevice);

/****************************/
    hipMalloc(&s_u0, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_u1, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_v0, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_v1, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_w0, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_w1, nnz * nnx * nny * sizeof(float));

    hipMalloc(&s_P, nnz * nnx * nny * sizeof(float));

    hipMalloc(&s_px0, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_px1, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_py0, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_py1, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_pz0, nnz * nnx * nny * sizeof(float));
    hipMalloc(&s_pz1, nnz * nnx * nny * sizeof(float));

    hipMalloc(&coffx1, nnx * sizeof(float));
    hipMalloc(&coffx2, nnx * sizeof(float));
    hipMalloc(&coffy1, nny * sizeof(float));
    hipMalloc(&coffy2, nny * sizeof(float));
    hipMalloc(&coffz1, nnz * sizeof(float));
    hipMalloc(&coffz2, nnz * sizeof(float));
    hipMalloc(&acoffx1, nnx * sizeof(float));
    hipMalloc(&acoffx2, nnx * sizeof(float));
    hipMalloc(&acoffy1, nny * sizeof(float));
    hipMalloc(&acoffy2, nny * sizeof(float));
    hipMalloc(&acoffz1, nnz * sizeof(float));
    hipMalloc(&acoffz2, nnz * sizeof(float));

    hipMalloc(&shot_Dev, nx * ny * nt * sizeof(float));

    error=hipGetLastError();
    if(error != hipSuccess){
        printf("%s\n",hipGetErrorString(error));
    }
/******************************/
    check_gpu_error("Failed to allocate memory for variables!");

    get_d0 <<< 1, 1 >>> (dx, dy, dz, nnx, nny, nnz, npml, vp);
    initial_coffe <<< (nnx + 511) / 512, 512 >>> (dt, nx, coffx1, coffx2, acoffx1, acoffx2, npml);
    initial_coffe <<< (nny + 511) / 512, 512 >>> (dt, ny, coffy1, coffy2, acoffy1, acoffy2, npml);
    initial_coffe <<< (nnz + 511) / 512, 512 >>> (dt, nz, coffz1, coffz2, acoffz1, acoffz2, npml);


/**********IS Loop start*******/
    for (is = 0; is < ns; is++) {
        //  printf("---   IS=%3d  \n",is);

        hipMemset(s_u0, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_u1, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_v0, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_v1, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_w0, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_w1, 0, nnz * nnx * nny * sizeof(float));

        hipMemset(s_P, 0, nnz * nnx * nny * sizeof(float));

        hipMemset(s_px0, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_px1, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_py0, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_py1, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_pz0, 0, nnz * nnx * nny * sizeof(float));
        hipMemset(s_pz1, 0, nnz * nnx * nny * sizeof(float));

        hipMemset(shot_Dev, 0, nt * nx * ny * sizeof(float));

        for (it = 0, t = dt; it < nt; it++, t += dt) {
            //if (it % snap_interval == 0)printf("it===%d\n", is, it);
            add_source <<< 1, 1 >>>
                               (pfac, fsx, fsy, zs, nx, ny, nz, nnx, nny, nnz, dt, t, favg, wtype, npml, is, dsx, dsy, s_P, nsx);
            hipDeviceSynchronize();
            update_vel <<< dimg, dimb >>> (nx, ny, nz, nnx, nny, nnz, npml, dt, dx, dy, dz,
                    s_u0, s_v0, s_w0, s_u1, s_v1, s_w1, s_P, coffx1, coffx2, coffy1, coffy2, coffz1, coffz2, density);
            hipDeviceSynchronize();
            update_stress <<< dimg, dimb >>>
                                     (nx, ny, nz, nnx, nny, nnz, dt, dx, dy, dz, s_u1, s_v1, s_w1, s_P, vp, density, npml,
                                             s_px1, s_px0, s_py1, s_py0, s_pz1, s_pz0,
                                             acoffx1, acoffx2, acoffy1, acoffy2, acoffz1, acoffz2,
                                             fsx, dsx, fsy, dsy, zs, is, nsx);
            hipDeviceSynchronize();
            ptr = s_u0;
            s_u0 = s_u1;
            s_u1 = ptr;
            
            ptr = s_v0;
            s_v0 = s_v1;
            s_v1 = ptr;
            
            ptr = s_w0;
            s_w0 = s_w1;
            s_w1 = ptr;
            
            ptr = s_px0;
            s_px0 = s_px1;
            s_px1 =ptr;
            
            ptr = s_py0;
            s_py0 = s_py1;
            s_py1 = ptr;
            
            ptr = s_pz0;
            s_pz0 = s_pz1;
            s_pz1 = ptr;

            shot_record <<< (nx * ny + 511) / 512, 512 >>> (nnx, nny, nnz, nx, ny, nz, npml, it, nt, s_P, shot_Dev);
            hipDeviceSynchronize();


            if (show_snapshot) {
                if(it % snap_interval == 0){
                    hipMemcpy(e, s_P, nnz*nnx*nny*sizeof(float), hipMemcpyDeviceToHost);
                    strcpy(snapname,FNsnap);
                    sprintf(snapid,"ishot_%d",is);
                    strcat(snapname,snapid);
                    sprintf(snapid,"it_%d",it);
                    strcat(snapname,snapid);
                    strcat(snapname,".bin");
                    if((fpsnap=fopen(snapname,"wb"))==NULL){
                        printf("cannot write snapfile\n");
                    }
                    window3d(v, e, nz, nx, ny, nnz, nnx, npml);
                    fwrite(v, sizeof(float), nx * nz * ny, fpsnap);
                    fclose(fpsnap);
                }
            }
        }//it loop end
        if (cut_directwave) {
            mute_directwave<<<Xdimg,dimb>>>(nx,ny,nt,dt,favg,dx,dy,dz,fsx,fsy,dsx,dsy,zs,is,vp,shot_Dev,60,nsx);
        }
        hipMemcpy(shot_Hos, shot_Dev, nt * nx * ny * sizeof(float), hipMemcpyDeviceToHost);
        fseek(fpshot, is * nt * nx * ny * sizeof(float), 0);
        for(iy=0;iy<ny;iy+=int(dgy/dy)){
            for(ix=0;ix<nx;ix+=int(dgx/dx)){
                  for(it=0;it<nt;it+=int(dgt/dt)){
                        fwrite(&shot_Hos[iy*nx*nt+ix*nt+it],sizeof(float),1,fpshot);
                  }
            }
        }

    }//is loop end

/*********IS Loop end*********/
    //printf("---   The forward is over    \n");
    //printf("---   Complete!!!!!!!!! \n");
    //printf("total %d shots: %f (s)\n", ns, ((float) (end - start)) / CLOCKS_PER_SEC);



/***********close************/
    fclose(fpshot);
/***********free*************/


    hipFree(coffx1);
    hipFree(coffx2);
    hipFree(coffy1);
    hipFree(coffy2);
    hipFree(coffz1);
    hipFree(coffz2);
    hipFree(acoffx1);
    hipFree(acoffx2);
    hipFree(acoffy1);
    hipFree(acoffy2);
    hipFree(acoffz1);
    hipFree(acoffz2);
    
    error=hipGetLastError();
    if(error != hipSuccess){
        printf("1%s\n",hipGetErrorString(error));
    }
    hipFree(s_u0);
    hipFree(s_u1);
    hipFree(s_v0);
    hipFree(s_v1);
    hipFree(s_w0);
    hipFree(s_w1);
    error=hipGetLastError();
    if(error != hipSuccess){
        printf("2%s\n",hipGetErrorString(error));
    }
    hipFree(s_P);
    error=hipGetLastError();
    if(error != hipSuccess){
        printf("3%s\n",hipGetErrorString(error));
    }
    hipFree(s_px0);
    hipFree(s_px1);
    hipFree(s_py0);
    hipFree(s_py1);
    hipFree(s_pz0);
    hipFree(s_pz1);
    error=hipGetLastError();
    if(error != hipSuccess){
        printf("4%s\n",hipGetErrorString(error));
    }
    hipFree(shot_Dev);
    error=hipGetLastError();
    if(error != hipSuccess){
        printf("5%s\n",hipGetErrorString(error));
    }
    hipFree(vp);
    hipFree(density);
    error=hipGetLastError();
    if(error != hipSuccess){
        printf("6%s\n",hipGetErrorString(error));
    }
/***************host free*****************/
    free(v);
    free(rho);
    free(shot_Hos);
}

